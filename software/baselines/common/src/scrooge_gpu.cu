#include <string>
#include <vector>
#include <iostream>
#include <fstream>

#include "genasm_cpu.hpp"
#include "genasm_gpu.hpp"
#include "util.hpp"

using namespace std;


void gpu_pairwise(std::vector<std::string> texts, std::vector<std::string> queries){
    vector<Alignment_t> alignments = genasm_gpu::align_all(texts, queries);

    // for(Alignment_t &aln : alignments){
    //     cout << "edit_distance:" << aln.edit_distance << " ";
    //     cout << "cigar:" << aln.cigar << endl;
    // }
}

int main(int argc, char *argv[]){
    genasm_cpu::enabled_algorithm_log = false;
    genasm_gpu::enabled_algorithm_log = true;
    
    std::ifstream rf (argv[1]);
    std::ifstream qf (argv[2]);

    std::vector<std::string> ref, query;
    std::string r,q;

    while(true){
        std::getline(rf, r);
        std::getline(qf, q);
        if (rf.eof()||qf.eof()) break;

        std::getline(rf, r);
        std::getline(qf, q);
        ref.push_back(r);
        query.push_back(q);
    }
    gpu_pairwise(ref, query);
    //cpu_string_pairs_example();
    //cpu_mapping_example();
    //gpu_mapping_example();
    // gpu_string_pairs_example();
}
